#include "hip/hip_runtime.h"
/**************************************************************************
*
*     set up GPU for processing
*
**************************************************************************/

#include "gpu_main.h"
#include <stdio.h>
#include <hip/hip_texture_types.h>

texture<float, 2> tex;

GPU_Palette openPalette(int theWidth, int theHeight)
{
  unsigned long theSize = theWidth * theHeight;
  unsigned long memSize = theSize * sizeof(float);

  float* redmap = (float*) malloc(memSize);
  float* greenmap = (float*) malloc(memSize);
  float* bluemap = (float*) malloc(memSize);

  for(int i = 0; i < theSize; i++){
    bluemap[i] 	= .0;
    greenmap[i] = .0;
    redmap[i]   = .0;
  }

  GPU_Palette P1 = initGPUPalette(theWidth, theHeight);

  hipMemcpy(P1.red, redmap, memSize, cH2D);
  hipMemcpy(P1.green, greenmap, memSize, cH2D);
  hipMemcpy(P1.blue, bluemap, memSize, cH2D);

  free(redmap);
  free(greenmap);
  free(bluemap);

  return P1;
}

/******************************************************************************/
GPU_Palette initGPUPalette(unsigned int imageWidth, unsigned int imageHeight)
{
  GPU_Palette X;

  X.gThreads.x = 32;  // 32 x 32 = 1024 threads per block
  X.gThreads.y = 32;
  X.gThreads.z = 1;
  X.gBlocks.x = ceil(imageWidth/32);  // however many blocks needed for image
  X.gBlocks.y = ceil(imageHeight/32);
  X.gBlocks.z = 1;

  X.palette_width = imageWidth;       // save this info
  X.palette_height = imageHeight;
  X.num_pixels = imageWidth * imageHeight;

  // allocate memory on GPU corresponding to pixel colors:
  hipError_t err;
  err = hipMalloc((void**) &X.red, X.num_pixels * sizeof(float));
  if(err != hipSuccess){
    printf("cuda error allocating red = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }
  hipMalloc((void**) &X.green, X.num_pixels * sizeof(float)); // g
  if(err != hipSuccess){
    printf("cuda error allocating green = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }
  hipMalloc((void**) &X.blue, X.num_pixels * sizeof(float));  // b
  if(err != hipSuccess){
    printf("cuda error allocating blue = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }

  hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
  hipBindTexture2D(NULL, tex, X.blue, desc, X.palette_width, 
                    X.palette_width, sizeof(float) * X.palette_width);

  return X;
}

/******************************************************************************/
void freeGPUPalette(GPU_Palette* P)
{
  hipUnbindTexture(tex);

  hipFree(P->red);
  hipFree(P->green);
  hipFree(P->blue);
}


/******************************************************************************/
int updatePalette(GPU_Palette* P, const Points& Points)
{
  for (Point Point : Points.points) {
    updateReds   <<< P->gBlocks, P->gThreads >>> (P->red,   Point);
    updateGreens <<< P->gBlocks, P->gThreads >>> (P->green, Point);
    updateBlues  <<< P->gBlocks, P->gThreads >>> (P->blue,  Point);
  }
  return 0;
}

/******************************************************************************/
__global__ void updateReds(float* red, Point Point){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int vecIdx = x + (y * blockDim.x * gridDim.x);

  int pointSize = round(Point.z*0.65);
          // x - xIdx+5 ???
  if( (powf((x+5 - Point.xIdx), 2) + powf((y+5 - Point.yIdx), 2)) < powf(pointSize, 2)) 
    red[vecIdx] = Point.red;
  else {
    if (Point.color_heatTransfer == 0) {
      float t, l, c, r, b;
      float speed = 0.1;
      t = tex2D(tex,x,y-pointSize/2);       
      l = tex2D(tex,x-pointSize/2,y);        
      c = tex2D(tex,x,y);        
      r = tex2D(tex,x+pointSize/2,y);        
      b = tex2D(tex,x,y+pointSize/2);      
      red[vecIdx] = c + speed * (t + b + r + l - 4 * c);
    }
    // red[vecIdx] *= Point.red_fadeScale;
  }
}

/******************************************************************************/
__global__ void updateGreens(float* green, Point Point){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int vecIdx = x + (y * blockDim.x * gridDim.x);

  int pointSize = round(Point.z*0.65);

  if( (powf((x+5 - Point.xIdx), 2) + powf((y+5 - Point.yIdx), 2)) < powf(pointSize, 2)) 
    green[vecIdx] = Point.green;
  else {
    if (Point.color_heatTransfer == 1) {
      float t, l, c, r, b;
      float speed = 0.45;
      t = tex2D(tex,x,y-pointSize/2);       
      l = tex2D(tex,x-pointSize/2,y);        
      c = tex2D(tex,x,y);        
      r = tex2D(tex,x+pointSize/2,y);        
      b = tex2D(tex,x,y+pointSize/2);      
      green[vecIdx] = c + speed * (t + b + r + l - 4 * c);
    }
    // green[vecIdx] *= Point.green_fadeScale;
  }
}

/******************************************************************************/
__global__ void updateBlues(float* blue, Point Point){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int vecIdx = x + (y * blockDim.x * gridDim.x);

  int pointSize = round(Point.z*0.65);

  if( (powf((x+5 - Point.xIdx), 2) + powf((y+5 - Point.yIdx), 2)) < powf(pointSize, 2)) 
    blue[vecIdx] = Point.blue;
  else {
    if (Point.color_heatTransfer == 2) {    
      float t, l, c, r, b;
      float speed = 0.25;
      t = tex2D(tex,x,y-pointSize/2);       
      l = tex2D(tex,x-pointSize/2,y);        
      c = tex2D(tex,x,y);        
      r = tex2D(tex,x+pointSize/2,y);        
      b = tex2D(tex,x,y+pointSize/2);      
      blue[vecIdx] = c + speed * (t + b + r + l - 4 * c);
    }
    // blue[vecIdx] *= Point.blue_fadeScale;
  }
}
/******************************************************************************/
