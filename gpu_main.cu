#include "hip/hip_runtime.h"
#include "gpu_main.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BackgroundRed 0.0f
#define BackgroundGreen 0.0f
#define BackgroundBlue 0.0f

#define AttractorRed 0.5f
#define AttractorGreen 0.5f
#define AttractorBlue 0.0f

#define zInitialSize 5
#define zScale 0.4f
#define FadeSpeed 0.01f
#define HeatTransferSpeed 0.04f




texture<float, 2> texRed;
texture<float, 2> texGreen;
texture<float, 2> texBlue;

GPU_Palette initGPUPalette(unsigned int imageWidth, unsigned int imageHeight) {
    GPU_Palette X;

    X.gThreads.x = 32; // 32 x 32 = 1024 threads per block
    X.gThreads.y = 32;
    X.gThreads.z = 1;
    X.gBlocks.x = ceil(imageWidth / 32); // however many blocks ng++ -w -c interface.cpp $(F1) $(F2) $(F3) $(F4)eeded for image
    X.gBlocks.y = ceil(imageHeight / 32);
    X.gBlocks.z = 1;

    X.palette_width = imageWidth; // save this info
    X.palette_height = imageHeight;
    X.num_pixels = imageWidth * imageHeight;

    // allocate memory on GPU corresponding to pixel colors:
    hipError_t err;
    err = hipMalloc((void**)&X.red, X.num_pixels * sizeof(float));
    if (err != hipSuccess) {
        printf("cuda error allocating red = %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipMalloc((void**)&X.green, X.num_pixels * sizeof(float)); // g
    if (err != hipSuccess) {
        printf("cuda error allocating green = %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipMalloc((void**)&X.blue, X.num_pixels * sizeof(float)); // b
    if (err != hipSuccess) {
        printf("cuda error allocating blue = %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

    hipBindTexture2D(NULL, texRed, X.red, desc, imageWidth, imageHeight, sizeof(float) * imageWidth);
    hipBindTexture2D(NULL, texGreen, X.red, desc, imageWidth, imageHeight, sizeof(float) * imageWidth);
    hipBindTexture2D(NULL, texBlue, X.red, desc, imageWidth, imageHeight, sizeof(float) * imageWidth);

    return X;
}

void freeGPUPalette(GPU_Palette* P) {
    hipUnbindTexture(texRed);
    hipUnbindTexture(texGreen);
    hipUnbindTexture(texBlue);

    hipFree(P->red);
    hipFree(P->green);
    hipFree(P->blue);
}


int updatePalette(GPU_Palette* P, int xIdx, int yIdx, float zIdx, int index) {

    updateReds<<<P->gBlocks, P->gThreads>>>(P->red, xIdx, yIdx, zIdx, index);
    updateGreens<<<P->gBlocks, P->gThreads>>>(P->green, xIdx, yIdx, zIdx,index);
    updateBlues<<<P->gBlocks, P->gThreads>>>(P->blue, xIdx, yIdx, zIdx,index);

    return 0;
}

__global__ void updateReds(float* red, int xIdx, int yIdx, float zIdx, int index) {

    float r[5] = {0.9f, 0.9f, 0.9f, 0.9f, 0.9f}; 

    float size = zInitialSize + zIdx * zScale;
    int x = threadIdx.x + (blockIdx.x * blockDim.x);
    int y = threadIdx.y + (blockIdx.y * blockDim.y);
    int vecIdx = x + (y * blockDim.x * gridDim.x);

    float top, left, center, right, bot;

    top = tex2D(texRed, x, y + 1);
    left = tex2D(texRed, x - 1, y);
    center = tex2D(texRed, x, y);
    right = tex2D(texRed, x + 1, y);
    bot = tex2D(texRed, x, y - 1);

    if (sqrtf(powf((x - xIdx), 2) + powf((y - yIdx), 2)) < size) {

        red[vecIdx] = r[index];

    } else {

        float heat_average = (top + bot + right + left + center) / (5 - HeatTransferSpeed);

        if (heat_average >= r[index]) {
            red[vecIdx] = r[index] / 2;
        } else {
            red[vecIdx] = heat_average;
        }

        red[vecIdx] -= FadeSpeed * red[vecIdx];

        if (red[vecIdx] < BackgroundRed)
            red[vecIdx] = BackgroundRed;
        if (red[vecIdx] > r[index])
            red[vecIdx] = r[index];
    }
}

__global__ void updateGreens(float* green, int xIdx, int yIdx, float zIdx, int index) {

    const float g[5] = {0.9f, 0.0f, 0.9f, 0.0f, 0.9f}; 

    float size = zInitialSize + zIdx * zScale;
    int x = threadIdx.x + (blockIdx.x * blockDim.x);
    int y = threadIdx.y + (blockIdx.y * blockDim.y);
    int vecIdx = x + (y * blockDim.x * gridDim.x);

    float top, left, center, right, bot;

    top = tex2D(texRed, x, y + 1);
    left = tex2D(texRed, x - 1, y);
    center = tex2D(texRed, x, y);
    right = tex2D(texRed, x + 1, y);
    bot = tex2D(texRed, x, y - 1);

    if (sqrtf(powf((x - xIdx), 2) + powf((y - yIdx), 2)) < size) {

        green[vecIdx] = g[index];

    } else {

        float heat_average = (top + bot + right + left + center) / (5 - HeatTransferSpeed);

        if (heat_average >= g[index]) {
            green[vecIdx] = g[index] / 2;
        } else {
            green[vecIdx] = heat_average;
        }

        green[vecIdx] -= FadeSpeed * green[vecIdx];

        if (green[vecIdx] < BackgroundGreen)
            green[vecIdx] = BackgroundGreen;
        if (green[vecIdx] > g[index])
            green[vecIdx] = g[index];

    }
}

__global__ void updateBlues(float* blue, int xIdx, int yIdx, float zIdx, int index) {

    const float b[5] = {0.9f, 0.0f, 0.9f, 0.0f, 0.9f}; 

    float size = zInitialSize + zIdx * zScale;
    int x = threadIdx.x + (blockIdx.x * blockDim.x);
    int y = threadIdx.y + (blockIdx.y * blockDim.y);
    int vecIdx = x + (y * blockDim.x * gridDim.x);

    float top, left, center, right, bot;

    top = tex2D(texRed, x, y + 1);
    left = tex2D(texRed, x - 1, y);
    center = tex2D(texRed, x, y);
    right = tex2D(texRed, x + 1, y);
    bot = tex2D(texRed, x, y - 1);

    if (sqrtf(powf((x - xIdx), 2) + powf((y - yIdx), 2)) < size) {

        blue[vecIdx] = b[index];

    } else {

        float heat_average = (top + bot + right + left + center) / (5 - HeatTransferSpeed);

        if (heat_average >= b[index]) {
          blue[vecIdx] = b[index] / 2;
        } else {
          blue[vecIdx] = heat_average;
        }

        blue[vecIdx] -= FadeSpeed * blue[vecIdx];

        if (blue[vecIdx] < BackgroundBlue)
            blue[vecIdx] = BackgroundBlue;
        if (blue[vecIdx] > b[index])
            blue[vecIdx] = b[index];
    }
}

