#include "hip/hip_runtime.h"
/**************************************************************************
*
*     set up GPU for processing
*
**************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "gpu_main.h"
#include <hip/hip_runtime.h>

#define gScalar 0.2
texture<float, 2, hipReadModeElementType> texBlue;

/******************************************************************************/
GPU_Palette initGPUPalette(unsigned int imageWidth, unsigned int imageHeight)
{
  GPU_Palette X;

  X.gThreads.x = 32;  // 32 x 32 = 1024 threads per block
  X.gThreads.y = 32;
  X.gThreads.z = 1;
  X.gBlocks.x = ceil(imageWidth/32);  // however many blocks needed for image
  X.gBlocks.y = ceil(imageHeight/32);
  X.gBlocks.z = 1;

  X.palette_width = imageWidth;       // save this info
  X.palette_height = imageHeight;
  X.num_pixels = imageWidth * imageHeight;

  // allocate memory on GPU corresponding to pixel colors:
  hipError_t err;
  err = hipMalloc((void**) &X.red, X.num_pixels * sizeof(float));
  if(err != hipSuccess){
    printf("cuda error allocating red = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }
  hipMalloc((void**) &X.green, X.num_pixels * sizeof(float)); // g
  if(err != hipSuccess){
    printf("cuda error allocating green = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }
  hipMalloc((void**) &X.blue, X.num_pixels * sizeof(float));  // b
  if(err != hipSuccess){
    printf("cuda error allocating blue = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }

  return X;
}

/******************************************************************************/
void freeGPUPalette(GPU_Palette* P)
{
  hipFree(P->red);
  hipFree(P->green);
  hipFree(P->blue);
}


/******************************************************************************/
int updatePalette(GPU_Palette* P, int xIdx, int yIdx, float z)
{

  updateReds <<< P->gBlocks, P->gThreads >>> (P->red, xIdx, yIdx,z);
  updateGreens <<< P->gBlocks, P->gThreads >>> (P->green, xIdx, yIdx,z);
	updateBlues <<< P->gBlocks, P->gThreads >>> (P->blue, xIdx, yIdx,z);

  return 0;
}

/******************************************************************************/
__global__ void updateReds(float* red, int xIdx, int yIdx, float z){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int vecIdx = x + (y * blockDim.x * gridDim.x);
  int pS = (int)(5 + (z * gScalar));

  if ((xIdx < x + pS) && (xIdx > x - pS) && (yIdx < y + pS) && (yIdx > y - pS)) {
   red[vecIdx] = 1.0;
 } else {
   red[vecIdx] *= .98;
 }
}

/******************************************************************************/
__global__ void updateGreens(float* green, int xIdx, int yIdx,float z){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int vecIdx = x + (y * blockDim.x * gridDim.x);
  int pS = (int)(5 + (z * gScalar));
  if ((xIdx < x + pS) && (xIdx > x - pS) && (yIdx < y + pS) && (yIdx > y - pS)) {
    green[vecIdx] = 1.0;
  } else {
    green[vecIdx] *= .90;
  }
}

/******************************************************************************/
__global__ void updateBlues(float* blue, int xIdx, int yIdx, float z){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int vecIdx = x + (y * blockDim.x * gridDim.x);
  int pS = (int)(5 + (z * gScalar));

  if ((xIdx < x + pS) && (xIdx > x - pS) && (yIdx < y + pS) && (yIdx > y - pS)) {
    blue[vecIdx] = 1.0;
  } else {
    float acc = 0.0;
    for (int i = -5;i <= 5;i++) {
      for (int j = -5;j <= 5;j++) {
        acc += tex2D(texBlue, x + i, y + j);
      }
    }
    acc /= 121.0;
    blue[vecIdx] = acc;
  }
}

/******************************************************************************/
