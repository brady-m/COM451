#include "hip/hip_runtime.h"
/**************************************************************************
*
*     set up GPU for processing
*
**************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "gpu_main.h"
#include <hip/hip_runtime.h>
#include <math.h> 
#include "draw.h"


#define gScalar 0.2

texture<float, 2> texRed;
texture<float, 2> texGreen;
texture<float, 2> texBlue;

/******************************************************************************/
GPU_Palette initGPUPalette(unsigned int imageWidth, unsigned int imageHeight)
{
  GPU_Palette X;

  X.gThreads.x = 32;  // 32 x 32 = 1024 threads per block
  X.gThreads.y = 32;
  X.gThreads.z = 1;
  X.gBlocks.x = ceil(imageWidth/32);  // however many blocks needed for image
  X.gBlocks.y = ceil(imageHeight/32);
  X.gBlocks.z = 1;

  X.palette_width = imageWidth;       // save this info
  X.palette_height = imageHeight;
  X.num_pixels = imageWidth * imageHeight;

  // allocate memory on GPU corresponding to pixel colors:
  hipError_t err;
  err = hipMalloc((void**) &X.red, X.num_pixels * sizeof(float));
  if(err != hipSuccess){
    printf("cuda error allocating red = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }
  hipMalloc((void**) &X.green, X.num_pixels * sizeof(float)); // g
  if(err != hipSuccess){
    printf("cuda error allocating green = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }
  hipMalloc((void**) &X.blue, X.num_pixels * sizeof(float));  // b
  if(err != hipSuccess){
    printf("cuda error allocating blue = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }


    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
  hipBindTexture2D(NULL, texRed, X.red, desc, X.palette_width, 
                    X.palette_width, sizeof(float) * X.palette_width);
  hipBindTexture2D(NULL, texGreen, X.green, desc, X.palette_width, 
                    X.palette_width, sizeof(float) * X.palette_width);
  hipBindTexture2D(NULL, texBlue, X.blue, desc, X.palette_width, 
                    X.palette_width, sizeof(float) * X.palette_width);


return X;
}

/******************************************************************************/
void freeGPUPalette(GPU_Palette* P)
{
  hipFree(P->red);
  hipFree(P->green);
  hipFree(P->blue);
}


/******************************************************************************/
int updatePalette(GPU_Palette* P, APoint (&points)[5])
{
  for (int i = 0;i < 5;i++) {
    updateReds <<< P->gBlocks, P->gThreads >>> (P->red, points[i].xIdx, points[i].yIdx, points[i].z, points[i].color_heatTransfer);
    updateGreens <<< P->gBlocks, P->gThreads >>> (P->green, points[i].xIdx, points[i].yIdx, points[i].z, points[i].color_heatTransfer);
  	updateBlues <<< P->gBlocks, P->gThreads >>> (P->blue, points[i].xIdx, points[i].yIdx, points[i].z, points[i].color_heatTransfer);
  }
  return 0;
}

/******************************************************************************/
__global__ void updateReds(float* red, int xIdx, int yIdx, float z, double colorTransfer){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int vecIdx = x + (y * blockDim.x * gridDim.x);

///////
  int pointSize = round(z*0.65);
  // x - xIdx+5 ???
  if( (powf((x+5 - xIdx), 2) + powf((y+5 - yIdx), 2)) < powf(pointSize, 2)) 
    red[vecIdx] = 1;
  else {
    red[vecIdx] *= 0.99;
  }
/////

//  if(xIdx == x && yIdx == y) red[vecIdx] = 1.0;
}

/******************************************************************************/
__global__ void updateGreens(float* green, int xIdx, int yIdx, float z, double colorTransfer){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int vecIdx = x + (y * blockDim.x * gridDim.x);

  int pS = round(z*0.65);
  if ((powf((x+5 - xIdx), 2) + powf((y+5 -yIdx), 2)) < powf(pS, 2)) {
      green[vecIdx] = 1.0;
    } else {
      green[vecIdx] *= .90;
    }


//  if(xIdx == x && yIdx == y) green[vecIdx] = 1.0;
}

/******************************************************************************/
__global__ void updateBlues(float* blue, int xIdx, int yIdx, float z, double colorTransfer){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int vecIdx = x + (y * blockDim.x * gridDim.x);

  int pS = round(z*0.65);
  if ((powf((x+5 - xIdx), 2) + powf((y+5 -yIdx), 2)) < powf(pS, 2)) {
      blue[vecIdx] = 1.0;
    } else {
      blue[vecIdx] *= .90;
    }

  //if(xIdx == x && yIdx == y) blue[vecIdx] = 1.0;
}

/******************************************************************************/
